#include "hip/hip_runtime.h"
/*
 * GStreamer
 * Copyright (C) <1999> Erik Walthinsen <omega@cse.ogi.edu>
 * Copyright (C) <2003> David Schleef <ds@schleef.org>
 * Copyright (C) <2012> Mikhail Durnev <mdurnev@rhonda.ru>
 * Copyright (C) <2014> Mikhail Durnev <mikhail_durnev@mentor.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 *
 * Alternatively, the contents of this file may be used under the
 * GNU Lesser General Public License Version 2.1 (the "LGPL"), in
 * which case the following provisions apply instead of the ones
 * mentioned above:
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Library General Public
 * License as published by the Free Software Foundation; either
 * version 2 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Library General Public License for more details.
 *
 * You should have received a copy of the GNU Library General Public
 * License along with this library; if not, write to the
 * Free Software Foundation, Inc., 59 Temple Place - Suite 330,
 * Boston, MA 02111-1307, USA.
 */

/**
 * SECTION:element-plugin
 *
 * FIXME:Describe plugin here.
 *
 * <refsect2>
 * <title>Example launch line</title>
 * |[
 * gst-launch -v -m videotestsrc ! plugin ! autovideosink
 * ]|
 * </refsect2>
 */
 
#ifdef HAVE_CONFIG_H
#include "../../common/config.h"
#endif

#include <gst/gst.h>
#include <gst/video/video.h>
#include <gst/video/gstvideofilter.h>
#include <string.h>

typedef unsigned int uint32_t;

#define PLAGIN_NAME "cudahost2gpu"
#define PLAGIN_SHORT_DESCRIPTION "cudahost2gpu Filter"

GST_DEBUG_CATEGORY_STATIC (gst_plugin_template_debug);
#define GST_CAT_DEFAULT gst_plugin_template_debug

typedef struct _GstCudahost2gpu GstCudahost2gpu;
typedef struct _GstCudahost2gpuClass GstCudahost2gpuClass;

#define GST_TYPE_PLUGIN_TEMPLATE \
  (gst_plugin_template_get_type())
#define GST_PLUGIN_TEMPLATE(obj) \
  (G_TYPE_CHECK_INSTANCE_CAST((obj),GST_TYPE_PLUGIN_TEMPLATE,GstCudahost2gpu))
#define GST_PLUGIN_TEMPLATE_CLASS(klass) \
  (G_TYPE_CHECK_CLASS_CAST((klass),GST_TYPE_PLUGIN_TEMPLATE,GstCudahost2gpuClass))
#define GST_IS_PLUGIN_TEMPLATE(obj) \
  (G_TYPE_CHECK_INSTANCE_TYPE((obj),GST_TYPE_PLUGIN_TEMPLATE))
#define GST_IS_PLUGIN_TEMPLATE_CLASS(klass) \
  (G_TYPE_CHECK_CLASS_TYPE((klass),GST_TYPE_PLUGIN_TEMPLATE))

struct _GstCudahost2gpu
{
  GstVideoFilter videofilter;

  gint width;
  gint height;
};

struct _GstCudahost2gpuClass
{
  GstVideoFilterClass parent_class;
};


enum
{
  /* FILL ME */
  LAST_SIGNAL
};

enum
{
  PROP_0,
};

/* debug category for fltering log messages
 */
#define DEBUG_INIT(bla) \
  GST_DEBUG_CATEGORY_INIT (gst_plugin_template_debug, PLAGIN_NAME, 0, PLAGIN_SHORT_DESCRIPTION);

GST_BOILERPLATE_FULL (GstCudahost2gpu, gst_plugin_template,
    GstVideoFilter, GST_TYPE_VIDEO_FILTER, DEBUG_INIT);


static void gst_plugin_template_set_property (GObject * object,
    guint prop_id, const GValue * value, GParamSpec * pspec);
static void gst_plugin_template_get_property (GObject * object,
    guint prop_id, GValue * value, GParamSpec * pspec);

static gboolean gst_plugin_template_set_caps (GstBaseTransform * bt,
    GstCaps * incaps, GstCaps * outcaps);
//static GstFlowReturn gst_plugin_template_filter (GstBaseTransform * bt,
//    GstBuffer * outbuf, GstBuffer * inbuf);
static GstFlowReturn
gst_plugin_template_filter_inplace (GstBaseTransform * base_transform,
    GstBuffer * buf);

#define ALLOWED_CAPS_STRING \
    GST_VIDEO_CAPS_BGRx

static GstStaticPadTemplate gst_video_filter_src_template =
GST_STATIC_PAD_TEMPLATE ("src",
    GST_PAD_SRC,
    GST_PAD_ALWAYS,
    GST_STATIC_CAPS (ALLOWED_CAPS_STRING)
    );

static GstStaticPadTemplate gst_video_filter_sink_template =
GST_STATIC_PAD_TEMPLATE ("sink",
    GST_PAD_SINK,
    GST_PAD_ALWAYS,
    GST_STATIC_CAPS (ALLOWED_CAPS_STRING)
    );

/* GObject vmethod implementations */

static void
gst_plugin_template_base_init (gpointer klass)
{
  GstElementClass *element_class = GST_ELEMENT_CLASS (klass);
  GstVideoFilterClass *videofilter_class = GST_VIDEO_FILTER_CLASS (klass);
  GstCaps *caps;

  gst_element_class_set_details_simple (element_class,
    PLAGIN_NAME,
    "Filter/Effect/Video",
    "Moves video frame to device memory",
    "Mikhail Durnev <mikhail_durnev@mentor.com>");

  gst_element_class_add_pad_template (element_class,
      gst_static_pad_template_get (&gst_video_filter_sink_template));
  gst_element_class_add_pad_template (element_class,
      gst_static_pad_template_get (&gst_video_filter_src_template));
}

static void
gst_plugin_template_class_init (GstCudahost2gpuClass * klass)
{
  GObjectClass *gobject_class;
  GstBaseTransformClass *btrans_class;
  GstVideoFilterClass *video_filter_class;

  gobject_class = (GObjectClass *) klass;
  btrans_class = (GstBaseTransformClass *) klass;
  video_filter_class = (GstVideoFilterClass *) klass;

  gobject_class->set_property = gst_plugin_template_set_property;
  gobject_class->get_property = gst_plugin_template_get_property;

  btrans_class->set_caps = gst_plugin_template_set_caps;
  btrans_class->transform = NULL;
  btrans_class->transform_ip = gst_plugin_template_filter_inplace;
}

static void
gst_plugin_template_init (GstCudahost2gpu * plugin_template,
    GstCudahost2gpuClass * g_class)
{
  GST_DEBUG ("init");
}

static void
gst_plugin_template_set_property (GObject * object, guint prop_id,
    const GValue * value, GParamSpec * pspec)
{
  GstCudahost2gpu *filter = GST_PLUGIN_TEMPLATE (object);

  GST_OBJECT_LOCK (filter);
  switch (prop_id) {
    default:
      G_OBJECT_WARN_INVALID_PROPERTY_ID (object, prop_id, pspec);
      break;
  }
  GST_OBJECT_UNLOCK (filter);
}

static void
gst_plugin_template_get_property (GObject * object, guint prop_id,
    GValue * value, GParamSpec * pspec)
{
  GstCudahost2gpu *filter = GST_PLUGIN_TEMPLATE (object);

  GST_OBJECT_LOCK (filter);
  switch (prop_id) {
    default:
      G_OBJECT_WARN_INVALID_PROPERTY_ID (object, prop_id, pspec);
      break;
  }
  GST_OBJECT_UNLOCK (filter);
}

static gboolean
gst_plugin_template_set_caps (GstBaseTransform * bt,
    GstCaps * incaps, GstCaps * outcaps)
{
  GstCudahost2gpu *plugin_template;
  GstStructure *structure = NULL;
  gboolean ret = FALSE;

  plugin_template = GST_PLUGIN_TEMPLATE (bt);

  structure = gst_caps_get_structure (incaps, 0);

  GST_OBJECT_LOCK (plugin_template);
  if (gst_structure_get_int (structure, "width", &plugin_template->width) &&
      gst_structure_get_int (structure, "height", &plugin_template->height)) {

    /* Check width and height and modify other plugin_template members accordingly */
    ret = TRUE;
  }
  GST_OBJECT_UNLOCK (plugin_template);

  return ret;
}

static GstFlowReturn
gst_plugin_template_filter_inplace (GstBaseTransform * base_transform,
    GstBuffer * buf)
{
  GstCudahost2gpu *plugin_template = GST_PLUGIN_TEMPLATE (base_transform);
  GstVideoFilter *videofilter = GST_VIDEO_FILTER (base_transform);

  gint width = plugin_template->width;
  gint height = plugin_template->height;
  gint stride = width * 4;

  uint32_t *in = (uint32_t *) GST_BUFFER_DATA (buf);
  unsigned long long *out = (unsigned long long *) GST_BUFFER_DATA (buf);

  /* Allocate device memory */
  void *dframe = NULL;
  void *dbuf = NULL;
  size_t pitch, pitch2;
  hipError_t stat = hipMallocPitch(&dframe, &pitch, stride, height);
  if (stat != hipSuccess)
  {
      return GST_FLOW_ERROR;
  }
  stat = hipMallocPitch(&dbuf, &pitch2, stride, height);
  if (stat != hipSuccess)
  {
      hipFree(dframe);
      return GST_FLOW_ERROR;
  }

  /* Copy video buffer to the device */
  stat = hipMemcpy2D(dframe, pitch, (const void*)in, stride, stride, height, hipMemcpyHostToDevice);
  if (stat != hipSuccess)
  {
      hipFree(dframe);
      hipFree(dbuf);
      return GST_FLOW_ERROR;
  }

  /* Create texture object */
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = dframe;
  resDesc.res.pitch2D.desc = hipCreateChannelDesc<uchar4>();
  resDesc.res.pitch2D.pitchInBytes = pitch;
  resDesc.res.pitch2D.width = stride;
  resDesc.res.pitch2D.height = height;

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipTextureObject_t tex = 0;
  stat = hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
  if (stat != hipSuccess)
  {
      GST_DEBUG("%s\n", hipGetErrorString(stat));
      hipFree(dframe);
      hipFree(dbuf);
      return GST_FLOW_ERROR;
  }

  resDesc.res.pitch2D.devPtr = dbuf;
  resDesc.res.pitch2D.pitchInBytes = pitch2;

  hipTextureObject_t tex2 = 0;
  stat = hipCreateTextureObject(&tex2, &resDesc, &texDesc, NULL);
  if (stat != hipSuccess)
  {
      GST_DEBUG("%s\n", hipGetErrorString(stat));
      hipDestroyTextureObject(tex);
      hipFree(dframe);
      hipFree(dbuf);
      return GST_FLOW_ERROR;
  }

  /*
   * out[0] - device pointer to the allocated memory
   * out[1] - pitch in bytes
   * out[2] - texture object
   * out[3] - device memory allocated for image processing
   * out[4] - pitch in bytes
   * out[5] - texture object
   */

  out[0] = (unsigned long long)dframe;
  out[1] = (unsigned long long)pitch;
  out[2] = (unsigned long long)tex;
  out[3] = (unsigned long long)dbuf;
  out[4] = (unsigned long long)pitch2;
  out[5] = (unsigned long long)tex2;

  //GST_DEBUG("0x%x %d %d 0x%x %d %d", (int)out[0], (int)out[1], (int)out[2], (int)out[3], (int)out[4], (int)out[5]);

  return GST_FLOW_OK;
}

static gboolean
plugin_init (GstPlugin * plugin)
{
  return gst_element_register (plugin, PLAGIN_NAME, GST_RANK_NONE,
      GST_TYPE_PLUGIN_TEMPLATE);
}

/* gstreamer looks for this structure to register plugins
 */
GST_PLUGIN_DEFINE (
    GST_VERSION_MAJOR,
    GST_VERSION_MINOR,
    PLAGIN_NAME,
    PLAGIN_SHORT_DESCRIPTION,
    plugin_init,
    VERSION, "LGPL",
    "GStreamer",
    "http://gstreamer.net/"
);
