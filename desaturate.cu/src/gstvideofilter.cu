#include "hip/hip_runtime.h"
/*
 * GStreamer
 * Copyright (C) <1999> Erik Walthinsen <omega@cse.ogi.edu>
 * Copyright (C) <2003> David Schleef <ds@schleef.org>
 * Copyright (C) <2012> Mikhail Durnev <mdurnev@rhonda.ru>
 * Copyright (C) <2014> Mikhail Durnev <mikhail_durnev@mentor.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 *
 * Alternatively, the contents of this file may be used under the
 * GNU Lesser General Public License Version 2.1 (the "LGPL"), in
 * which case the following provisions apply instead of the ones
 * mentioned above:
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Library General Public
 * License as published by the Free Software Foundation; either
 * version 2 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Library General Public License for more details.
 *
 * You should have received a copy of the GNU Library General Public
 * License along with this library; if not, write to the
 * Free Software Foundation, Inc., 59 Temple Place - Suite 330,
 * Boston, MA 02111-1307, USA.
 */

/**
 * SECTION:element-plugin
 *
 * FIXME:Describe plugin here.
 *
 * <refsect2>
 * <title>Example launch line</title>
 * |[
 * gst-launch -v -m videotestsrc ! plugin ! autovideosink
 * ]|
 * </refsect2>
 */
 
#ifdef HAVE_CONFIG_H
#include "../../common/config.h"
#endif

#include <gst/gst.h>
#include <gst/video/video.h>
#include <gst/video/gstvideofilter.h>
#include <string.h>

#define CUDA_CHECK_RETURN(value) {                                          \
    hipError_t stat = value;                                        \
    if (stat != hipSuccess) {                                       \
        GST_DEBUG("Error %s at line %d in file %s\n",                 \
                hipGetErrorString(stat), __LINE__, __FILE__);       \
    } }

typedef unsigned int uint32_t;

#define PLAGIN_NAME "cudadesaturate"
#define PLAGIN_SHORT_DESCRIPTION "CUDA desaturate Filter"

GST_DEBUG_CATEGORY_STATIC (gst_plugin_template_debug);
#define GST_CAT_DEFAULT gst_plugin_template_debug

typedef struct _GstPlugincudadesaturate GstPlugincudadesaturate;
typedef struct _GstPlugincudadesaturateClass GstPlugincudadesaturateClass;

#define GST_TYPE_PLUGIN_TEMPLATE \
  (gst_plugin_template_get_type())
#define GST_PLUGIN_TEMPLATE(obj) \
  (G_TYPE_CHECK_INSTANCE_CAST((obj),GST_TYPE_PLUGIN_TEMPLATE,GstPlugincudadesaturate))
#define GST_PLUGIN_TEMPLATE_CLASS(klass) \
  (G_TYPE_CHECK_CLASS_CAST((klass),GST_TYPE_PLUGIN_TEMPLATE,GstPlugincudadesaturateClass))
#define GST_IS_PLUGIN_TEMPLATE(obj) \
  (G_TYPE_CHECK_INSTANCE_TYPE((obj),GST_TYPE_PLUGIN_TEMPLATE))
#define GST_IS_PLUGIN_TEMPLATE_CLASS(klass) \
  (G_TYPE_CHECK_CLASS_TYPE((klass),GST_TYPE_PLUGIN_TEMPLATE))

struct _GstPlugincudadesaturate
{
  GstVideoFilter videofilter;

  gint width;
  gint height;

  gint saturation;
};

struct _GstPlugincudadesaturateClass
{
  GstVideoFilterClass parent_class;
};


enum
{
  /* FILL ME */
  LAST_SIGNAL
};

enum
{
  PROP_0,
  PROP_SATURATION
};

/* debug category for fltering log messages
 */
#define DEBUG_INIT(bla) \
  GST_DEBUG_CATEGORY_INIT (gst_plugin_template_debug, PLAGIN_NAME, 0, PLAGIN_SHORT_DESCRIPTION);

GST_BOILERPLATE_FULL (GstPlugincudadesaturate, gst_plugin_template,
    GstVideoFilter, GST_TYPE_VIDEO_FILTER, DEBUG_INIT);

static void gst_plugin_template_set_property (GObject * object,
    guint prop_id, const GValue * value, GParamSpec * pspec);
static void gst_plugin_template_get_property (GObject * object,
    guint prop_id, GValue * value, GParamSpec * pspec);
static void gst_plugin_template_finalize (GObject * object);

static gboolean gst_plugin_template_set_caps (GstBaseTransform * bt,
    GstCaps * incaps, GstCaps * outcaps);
//static GstFlowReturn gst_plugin_template_filter (GstBaseTransform * bt,
//    GstBuffer * outbuf, GstBuffer * inbuf);
static GstFlowReturn
gst_plugin_template_filter_inplace (GstBaseTransform * base_transform,
    GstBuffer * buf);

#define ALLOWED_CAPS_STRING \
    GST_VIDEO_CAPS_BGRx

static GstStaticPadTemplate gst_video_filter_src_template =
GST_STATIC_PAD_TEMPLATE ("src",
    GST_PAD_SRC,
    GST_PAD_ALWAYS,
    GST_STATIC_CAPS (ALLOWED_CAPS_STRING)
    );

static GstStaticPadTemplate gst_video_filter_sink_template =
GST_STATIC_PAD_TEMPLATE ("sink",
    GST_PAD_SINK,
    GST_PAD_ALWAYS,
    GST_STATIC_CAPS (ALLOWED_CAPS_STRING)
    );

/* GObject method implementations */

static void
gst_plugin_template_base_init (gpointer klass)
{
  GstElementClass *element_class = GST_ELEMENT_CLASS (klass);
  GstVideoFilterClass *videofilter_class = GST_VIDEO_FILTER_CLASS (klass);
  GstCaps *caps;

  gst_element_class_set_details_simple (element_class,
    PLAGIN_NAME,
    "Filter/Effect/Video",
    "Desaturate",
    "Mikhail Durnev <mikhail_durnev@mentor.com>");

  gst_element_class_add_pad_template (element_class,
      gst_static_pad_template_get (&gst_video_filter_sink_template));
  gst_element_class_add_pad_template (element_class,
      gst_static_pad_template_get (&gst_video_filter_src_template));
}

static void
gst_plugin_template_class_init (GstPlugincudadesaturateClass * klass)
{
  GObjectClass *gobject_class;
  GstBaseTransformClass *btrans_class;
  GstVideoFilterClass *video_filter_class;

  gobject_class = (GObjectClass *) klass;
  btrans_class = (GstBaseTransformClass *) klass;
  video_filter_class = (GstVideoFilterClass *) klass;

  gobject_class->set_property = gst_plugin_template_set_property;
  gobject_class->get_property = gst_plugin_template_get_property;
  gobject_class->finalize = gst_plugin_template_finalize;

  g_object_class_install_property (gobject_class, PROP_SATURATION,
      g_param_spec_int ("saturation", "Saturation", "Saturation = ",
          0, 100, 10, (GParamFlags)G_PARAM_READWRITE));

  btrans_class->set_caps = gst_plugin_template_set_caps;
  btrans_class->transform = NULL;
  btrans_class->transform_ip = gst_plugin_template_filter_inplace;
}

static void
gst_plugin_template_init (GstPlugincudadesaturate * plugin_template,
    GstPlugincudadesaturateClass * g_class)
{
  GST_DEBUG ("init");

  plugin_template->saturation = 10;
}

static void
gst_plugin_template_set_property (GObject * object, guint prop_id,
    const GValue * value, GParamSpec * pspec)
{
  GstPlugincudadesaturate *filter = GST_PLUGIN_TEMPLATE (object);

  GST_OBJECT_LOCK (filter);
  switch (prop_id) {
    case PROP_SATURATION:
        filter->saturation = g_value_get_int (value);
        GST_DEBUG("saturation = %d\n", filter->saturation);
        break;
    default:
      G_OBJECT_WARN_INVALID_PROPERTY_ID (object, prop_id, pspec);
      break;
  }
  GST_OBJECT_UNLOCK (filter);
}

static void
gst_plugin_template_get_property (GObject * object, guint prop_id,
    GValue * value, GParamSpec * pspec)
{
  GstPlugincudadesaturate *filter = GST_PLUGIN_TEMPLATE (object);

  GST_OBJECT_LOCK (filter);
  switch (prop_id) {
    case PROP_SATURATION:
        g_value_set_int (value, filter->saturation);
        break;
    default:
      G_OBJECT_WARN_INVALID_PROPERTY_ID (object, prop_id, pspec);
      break;
  }
  GST_OBJECT_UNLOCK (filter);
}

static void
gst_plugin_template_finalize (GObject * object)
{
  GstPlugincudadesaturate *filter = GST_PLUGIN_TEMPLATE (object);

  //G_OBJECT_CLASS (object)->finalize (object);

  GST_DEBUG("finalize");
}


static gboolean
gst_plugin_template_set_caps (GstBaseTransform * bt,
    GstCaps * incaps, GstCaps * outcaps)
{
  GstPlugincudadesaturate *plugin_template;
  GstStructure *structure = NULL;
  gboolean ret = FALSE;

  plugin_template = GST_PLUGIN_TEMPLATE (bt);

  structure = gst_caps_get_structure (incaps, 0);

  GST_OBJECT_LOCK (plugin_template);
  if (gst_structure_get_int (structure, "width", &plugin_template->width) &&
      gst_structure_get_int (structure, "height", &plugin_template->height))
  {
    /* Check width and height and modify other plugin_template members accordingly */
    ret = TRUE;

  }
  GST_OBJECT_UNLOCK (plugin_template);

  return ret;
}

__global__ void video_filter(hipTextureObject_t in, uchar4* out, size_t pitch, int width, int height,
                             float saturation)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    uchar4 v = tex2D<uchar4>(in, x, y);

    int m = (v.x + v.y + v.z) / 3;

    v.x = (v.x - m) * saturation + m;
    v.y = (v.y - m) * saturation + m;
    v.z = (v.z - m) * saturation + m;

    *(uchar4*)((char*)out + (pitch * y + x * sizeof(uchar4))) = v;
}

static GstFlowReturn
gst_plugin_template_filter_inplace (GstBaseTransform * base_transform,
    GstBuffer * buf)
{
  GstPlugincudadesaturate *plugin_template = GST_PLUGIN_TEMPLATE (base_transform);
  GstVideoFilter *videofilter = GST_VIDEO_FILTER (base_transform);

  gint width = plugin_template->width;
  gint height = plugin_template->height;

  unsigned long long *in = (unsigned long long *) GST_BUFFER_DATA (buf);
  /*
   * in[0] - device pointer to the allocated memory
   * in[1] - pitch in bytes
   * in[2] - texture object
   * in[3] - device memory allocated for image processing
   * in[4] - pitch in bytes
   * in[5] - texture object
   */

  dim3 dimBlock(16, 16);
  dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x,
               (height + dimBlock.y - 1) / dimBlock.y);

  video_filter<<<dimGrid, dimBlock>>>((hipTextureObject_t)in[2], (uchar4*)in[3], (size_t)in[4], width, height,
                                      (float)(plugin_template->saturation) / 100.0);

  CUDA_CHECK_RETURN(hipDeviceSynchronize());
  CUDA_CHECK_RETURN(hipGetLastError());

  // Swap buffers
  int i;
  for (i = 0; i < 3; i++)
  {
      unsigned long long x = in[i];
      in[i] = in[i + 3];
      in[i + 3] = x;
  }

  return GST_FLOW_OK;
}

static gboolean
plugin_init (GstPlugin * plugin)
{
  return gst_element_register (plugin, PLAGIN_NAME, GST_RANK_NONE,
      GST_TYPE_PLUGIN_TEMPLATE);
}

/* gstreamer looks for this structure to register plugins
 */
GST_PLUGIN_DEFINE (
    GST_VERSION_MAJOR,
    GST_VERSION_MINOR,
    PLAGIN_NAME,
    PLAGIN_SHORT_DESCRIPTION,
    plugin_init,
    VERSION, "LGPL",
    "GStreamer",
    "http://gstreamer.net/"
);
